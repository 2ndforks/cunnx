#include "hip/hip_runtime.h"
#define MINUS_LOG_THRESHOLD -18.42
#define SOFTMAXTREE_THREADS 32
#define SOFTMAXTREE_MAXCHILDREN 100


__global__ void cunnx_SoftMaxTree_updateOutput_kernel(
  float *output, float* logsoftOutput, 
  float *input, float* weight, float* bias, 
  float* target, float* childParent, float* parentChildren, 
  int nInput, int rootId)
{
  //__shared__ float input_buffer[nInput]; // constant might be faster
  __shared__ float buffer[SOFTMAXTREE_THREADS+1];
  __shared__ float linearOutput[SOFTMAXTREE_MAXCHILDREN];
  int tx = threadIdx.x;
  int i_step = blockDim.x;
  int k = blockIdx.x;
  float *input_k = input + k*nInput;
  float *nodeOutput, *nodeWeight, *nodeBias;
  float narrowsum = 0;
  int childId = (*(target+k)) - 1;
  int parentId, parentIdx, childIdx, nChildren;
  int nOutput;
  float *node;
  int n = 0;
  
  // zero buffer
  buffer[tx] = 0;
  
  __syncthreads();

  // loop through nodes
  while(1)
  {
    /* get next Node in Tree */
    node = childParent + childId*2;
    parentId = (int)node[0] - 1;
    childIdx = (int)node[1] - 1;
    
    node = parentChildren + parentId*2;
    parentIdx = (int)node[0] - 1;
    nChildren = (int)node[1];
    
    /* Linear */
    
    nodeWeight = weight + parentIdx*nInput;
    nodeBias = bias + parentIdx;
    
    // addmv (dot products)
    for (int j=0; j<nChildren; j++)
    {
      // multiply
      for (int i=tx; i<nInput; i+=i_step)
      {
        buffer[tx] += input_k[i]*nodeWeight[i*nInput + j];
      }
      // add (reduce)
      for (unsigned int stride = blockDim.x >> 1; stride > 0; stride >>= 1)
      {
        __syncthreads();
        if (tx < stride)
          buffer[tx] += buffer[tx+stride];
      }
      if (tx == 0) 
        linearOutput[j] = buffer[0] + nodeBias[j];
    }
    
    if (tx == 0)
      output[k] = linearOutput[0];
      
    n += nChildren;
    /* Break when root is reached */
    if (parentId == rootId) 
    {
      break;
    }
    childId = parentId;
  }
  //if (tx == 0)
  //  output[k] = narrowsum;
}

static int cunnx_SoftMaxTree_updateOutput(lua_State *L)
{ 
  THCudaTensor *input = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");  
  THCudaTensor *target = (THCudaTensor*)luaT_checkudata(L, 3, "torch.CudaTensor");  
  int inputSize = luaT_getfieldcheckint(L, 1, "inputSize");
  int rootId = luaT_getfieldcheckint(L, 1, "rootId") - 1;
  int maxFamily = (int)luaT_getfieldcheckint(L, 1, "maxFamily");
  
  THCudaTensor *childParent = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "childParent", "torch.CudaTensor");
  THCudaTensor *parentChildren = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "parentChildren", "torch.CudaTensor");
  
  THCudaTensor *linearOutput = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "_nodeBuffer", "torch.CudaTensor");
  THCudaTensor *logsoftOutput = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "_multiBuffer", "torch.CudaTensor");
  
  THCudaTensor *weight = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "weight", "torch.CudaTensor");
  THCudaTensor *bias = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "bias", "torch.CudaTensor");
  THCudaTensor *output = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "output", "torch.CudaTensor");

  
  luaL_argcheck(L, input->nDimension == 2, 2, "2D(batch mode) tensor expected");
  luaL_argcheck(L, input->size[1] == inputSize, 2, "invalid input size");  
  
  input = THCudaTensor_newContiguous(input);
  THCudaTensor_resize1d(output, input->size[0]);
  
  /* call cudakernel */
  dim3 blocks(input->size[0]); // each block is an example
  dim3 threads(SOFTMAXTREE_THREADS);
  cunnx_SoftMaxTree_updateOutput_kernel<<<blocks,threads>>>(
    THCudaTensor_data(output), THCudaTensor_data(logsoftOutput), 
    THCudaTensor_data(input), THCudaTensor_data(weight), 
    THCudaTensor_data(bias), THCudaTensor_data(target), 
    THCudaTensor_data(childParent), THCudaTensor_data(parentChildren), 
    input->size[1], rootId
  );
  printf("here2 %f\n", THCudaTensor_get1d(logsoftOutput, 0));
  hipError_t errcode = hipGetLastError();
  if(errcode != hipSuccess)
    THError(hipGetErrorString(errcode));
  
  printf("here3\n");
  THCudaTensor_free(input);
  return 1;
}

static int cunnx_SoftMaxTree_updateGradInput(lua_State *L)
{
  return 1;
}

static const struct luaL_Reg cunnx_SoftMaxTree__ [] = {
  {"SoftMaxTree_updateOutput", cunnx_SoftMaxTree_updateOutput},
  {"SoftMaxTree_updateGradInput", cunnx_SoftMaxTree_updateGradInput},
  {NULL, NULL}
};

static void cunnx_SoftMaxTree_init(lua_State *L)
{
  luaT_pushmetatable(L, "torch.CudaTensor");
  luaT_registeratname(L, cunnx_SoftMaxTree__, "nn");
  lua_pop(L,1);
}
