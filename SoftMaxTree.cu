
static int cunnx_SoftMaxTree_updateOutput(lua_State *L)
{ 
  THCudaTensor *input = luaT_checkudata(L, 2, "torch.CudaTensor");  
  THIntTensor *target = (THIntTensor*)luaT_checkudata(L, 3, "torch.IntTensor");  
  int inputSize = luaT_getfieldcheckint(L, 1, "inputSize");
  long rootId = (long)(luaT_getfieldcheckint(L, 1, "rootId") - 1);
  
  THIntTensor *childParent = (THIntTensor*)luaT_getfieldcheckudata(L, 1, "childParent", "torch.IntTensor");
  THIntTensor *parentChildren = (THIntTensor*)luaT_getfieldcheckudata(L, 1, "parentChildren", "torch.IntTensor");
  
  THCudaTensor *linearOutput = luaT_getfieldcheckudata(L, 1, "_linearOutput", "torch.CudaTensor");
  THCudaTensor *logsoftOutput = luaT_getfieldcheckudata(L, 1, "_logSoftMaxOutput", "torch.CudaTensor");
  
  THCudaTensor *weight = luaT_getfieldcheckudata(L, 1, "weight", "torch.CudaTensor");
  THCudaTensor *bias = luaT_getfieldcheckudata(L, 1, "bias", "torch.CudaTensor");
  THCudaTensor *output = luaT_getfieldcheckudata(L, 1, "output", "torch.CudaTensor");
  
  THIntTensor *node;
  THCudaTensor *nodeWeight, *nodeBias, *nodeOutput, *nodeInput, *nodeInter;
  real *input_data, *output_data;

  long i, d;
  long n = 0;
  
  luaL_argcheck(L, input->nDimension == 2, 2, "2D(batch mode) tensor expected");
  luaL_argcheck(L, input->size[1] == inputSize, 2, "invalid input size");

  node = THIntTensor_new();
  nodeWeight = THTensor_(new)();
  nodeBias = THCudaTensor_(new)();
  nodeOutput = THCudaTensor_(new)();
  nodeInput = THCudaTensor_(new)();
  nodeInter = THCudaTensor_(new)();
  
  
  hipError_t errcode = hipGetLastError();
  if(errcode != hipSuccess)
    THError(hipGetErrorString(errcode));

  THCudaTensor_free(input);
  return 1;
}

static int cunnx_SoftMaxTree_updateGradInput(lua_State *L)
{
  return 1;
}

static const struct luaL_Reg cunnx_SoftMax__ [] = {
  {"SoftMaxTree_updateOutput", cunnx_SoftMaxTree_updateOutput},
  {"SoftMaxTree_updateGradInput", cunnx_SoftMaxTree_updateGradInput},
  {NULL, NULL}
};

static void cunnx_SoftMaxTree_init(lua_State *L)
{
  luaT_pushmetatable(L, "torch.CudaTensor");
  luaT_registeratname(L, cunnx_SoftMaxTree__, "nn");
  lua_pop(L,1);
}
