#include "hip/hip_runtime.h"
#define MINUS_LOG_THRESHOLD -18.42
#define SOFTMAXTREE_THREADS 32
#define SOFTMAXTREE_MAXCHILDREN 10000


__global__ void cunnx_SoftMaxTree_updateOutput_kernel(
  float *output, float* logsoftOutput,
  float *input, float* weight, float* bias, 
  float *target, float* childParent, float* parentChildren, 
  int nInput, int rootId, int maxFamilyPath)
{
  //__shared__ float input_buffer[nInput]; // constant might be faster
  __shared__ float buffer[SOFTMAXTREE_THREADS+1];
  __shared__ float linearOutput[SOFTMAXTREE_MAXCHILDREN];
  int tx = threadIdx.x;
  int i_step = blockDim.x;
  int k = blockIdx.x;
  float *input_k = input + k*nInput;
  float *nodeOutput, *nodeWeight, *nodeBias;
  float narrowsum = 0;
  int childId = target[k] - 1;
  int parentId, parentIdx, childIdx, nChildren;
  int nOutput;
  float *node;
  int n = 0;

  // loop through nodes
  while(1)
  {
    /* get next Node in Tree */
    node = childParent + childId*2;
    parentId = (int)node[0] - 1;
    childIdx = (int)node[1] - 1;
    
    node = parentChildren + parentId*2;
    parentIdx = (int)node[0] - 1;
    nChildren = (int)node[1];
    
    /* Linear */
    
    nodeWeight = weight + parentIdx*nInput;
    nodeBias = bias + parentIdx;
    
    // addmv (dot products)
    for (int j=0; j<nChildren; j++)
    {
       // zero buffer
      buffer[tx] = 0;
      __syncthreads();
      
      // multiply
      for (int i=tx; i<nInput; i+=i_step)
      {
        buffer[tx] += input_k[i]*nodeWeight[j*nInput + i];
      }
      // add (reduce)
      for (unsigned int stride = blockDim.x >> 1; stride > 0; stride >>= 1)
      {
        __syncthreads();
        if (tx < stride)
          buffer[tx] += buffer[tx+stride];
      }
      if (tx == 0) 
        linearOutput[j] = buffer[0] + nodeBias[j];
    }
    
    __syncthreads();
    
    /* LogSoftMax */
    nodeOutput = logsoftOutput + maxFamilyPath*k + n;
    
    // max?
    buffer[tx] = -FLT_MAX;
    for (int i=tx; i<nChildren; i+=i_step)
    {
      float z = linearOutput[i];
      if(buffer[tx] < z)
        buffer[tx] = z;
    }

    __syncthreads();
    
    // reduce
    nOutput = blockDim.x;
    if (nChildren < nOutput)
      nOutput = nChildren;
    if (tx == 0)
    {
      float max_k = -FLT_MAX;
      for (int i=0; i<nOutput; i++)
      {
        if(max_k < buffer[i])
          max_k = buffer[i];
      }
      buffer[SOFTMAXTREE_THREADS] = max_k;
    }

    __syncthreads();
    
    // logadd?
    float max_k = buffer[SOFTMAXTREE_THREADS];
    buffer[tx] = 0;
    for (int i=tx; i<nOutput; i+=i_step)
      buffer[tx] += __expf(linearOutput[i]-max_k);

    __syncthreads();

    // reduce
    if (tx == 0)
    {
      float logsum_k = 0;
      for (int i=0; i<nOutput; i++)
        logsum_k += buffer[i];
      buffer[SOFTMAXTREE_THREADS] = max_k + __logf(logsum_k);
    }

    __syncthreads();

    // logsoftmax
    float logsum_k = buffer[SOFTMAXTREE_THREADS];
    for (int i=tx; i<nOutput; i+=i_step)
      nodeOutput[i] = linearOutput[i] - logsum_k;
      
    __syncthreads();
    
    /* Narrow + CAddTable (without log, would have been CMulTable) */
    if (tx == 0)
      narrowsum += nodeOutput[childIdx];
      
    n += nChildren;
    /* Break when root is reached */
    if (parentId == rootId) 
    {
      break;
    }
    childId = parentId;
  }
  if (tx == 0)
    output[k] = narrowsum;
}


static int cunnx_SoftMaxTree_updateOutput(lua_State *L)
{ 
  THCudaTensor *input = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");  
  THCudaTensor *target = (THCudaTensor*)luaT_checkudata(L, 3, "torch.CudaTensor");  
  int inputSize = luaT_getfieldcheckint(L, 1, "inputSize");
  int rootId = luaT_getfieldcheckint(L, 1, "rootId") - 1;
  int maxFamilyPath = (int)luaT_getfieldcheckint(L, 1, "maxFamilyPath");
  
  THCudaTensor *childParent = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "childParent", "torch.CudaTensor");
  THCudaTensor *parentChildren = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "parentChildren", "torch.CudaTensor");

  THCudaTensor *logsoftOutput = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "_multiBuffer", "torch.CudaTensor");
  
  THCudaTensor *weight = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "weight", "torch.CudaTensor");
  THCudaTensor *bias = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "bias", "torch.CudaTensor");
  THCudaTensor *output = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "output", "torch.CudaTensor");
  
  luaL_argcheck(L, input->nDimension == 2, 2, "2D(batch mode) tensor expected");
  luaL_argcheck(L, input->size[1] == inputSize, 2, "invalid input size");  
  
  input = THCudaTensor_newContiguous(input);
  THCudaTensor_resize1d(output, input->size[0]);
  
  /* call cudakernel */
  dim3 blocks(input->size[0]); // each block is an example
  dim3 threads(SOFTMAXTREE_THREADS);
  cunnx_SoftMaxTree_updateOutput_kernel<<<blocks,threads>>>(
    THCudaTensor_data(output), THCudaTensor_data(logsoftOutput), 
    THCudaTensor_data(input), THCudaTensor_data(weight), 
    THCudaTensor_data(bias), THCudaTensor_data(target), 
    THCudaTensor_data(childParent), THCudaTensor_data(parentChildren), 
    input->size[1], rootId, maxFamilyPath
  );
  
  hipError_t errcode = hipGetLastError();
  if(errcode != hipSuccess)
    THError(hipGetErrorString(errcode));
  
  THCudaTensor_free(input);
  return 1;
}


__global__ void cunnx_SoftMaxTree_updateGradInput_kernel(
  float *gradInput, float* logsoftOutput, float *gradOutput, float* weight,
  float *target, float* childParent, float* parentChildren, 
  int nInput, int rootId, int maxFamilyPath)
{
  //__shared__ float input_buffer[nInput]; // constant might be faster
  __shared__ float buffer[SOFTMAXTREE_THREADS];
  int tx = threadIdx.x;
  int i_step = blockDim.x;
  int k = blockIdx.x;
  float *gradInput_k = gradInput + k*nInput;
  float *nodeGrad, *nodeWeight;
  float grad = gradOutput[k];
  int childId = target[k] - 1;
  int parentId, parentIdx, childIdx, nChildren;
  float *node;
  int n = 0;
  
  // zero gradInputs (for accumulation)
  for (int i=tx; i<nInput; i+=i_step)
    gradInput_k[i] = 0;

  // loop through nodes
  while(1)
  {
    /* get next Node in Tree */
    node = childParent + childId*2;
    parentId = (int)node[0] - 1;
    childIdx = (int)node[1] - 1;
    
    node = parentChildren + parentId*2;
    parentIdx = (int)node[0] - 1;
    nChildren = (int)node[1];
    
    /* CAddTable + Narrow + LogSoftMax */
    // AKA linearGradOutput (we reuse the _multiBuffer Tensor)
    nodeGrad = logsoftOutput + maxFamilyPath*k + n; 

    for(int i = tx; i < nChildren; i+=i_step)
      nodeGrad[i] = -exp(nodeGrad[i])*grad;
    
    __syncthreads();
    if (tx == 0) // compare this to % childIdx
      nodeGrad[childIdx] += grad;

    /* Linear */
    nodeWeight = weight + parentIdx*nInput;
    
    // addmv (dot products)
    for (int i=tx; i<nInput; i+=i_step)
    {
     // zero buffer
      buffer[tx] = 0;
      
      for (int j=0; j<nChildren; j++)
      {
        // multiply
        buffer[tx] += nodeGrad[j]*nodeWeight[j*nInput + i];
      }
      // accumulate into global memory
      gradInput_k[i] += buffer[tx];
    }
    
    n += nChildren;
    /* Break when root is reached */
    if (parentId == rootId)
    {
      break;
    }
    childId = parentId;
  }
}

static int cunnx_SoftMaxTree_updateGradInput(lua_State *L)
{
  THCudaTensor *input = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");  
  THCudaTensor *gradOutput = (THCudaTensor*)luaT_checkudata(L, 3, "torch.CudaTensor");  
  THCudaTensor *target = (THCudaTensor*)luaT_checkudata(L, 4, "torch.CudaTensor");  
  int inputSize = luaT_getfieldcheckint(L, 1, "inputSize");
  int rootId = luaT_getfieldcheckint(L, 1, "rootId") - 1;
  int maxFamilyPath = (int)luaT_getfieldcheckint(L, 1, "maxFamilyPath");
  
  THCudaTensor *childParent = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "childParent", "torch.CudaTensor");
  THCudaTensor *parentChildren = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "parentChildren", "torch.CudaTensor");
  
  THCudaTensor *logsoftOutput = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "_multiBuffer", "torch.CudaTensor");
  
  THCudaTensor *weight = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "weight", "torch.CudaTensor");
  THCudaTensor *bias = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "bias", "torch.CudaTensor");
  THCudaTensor *gradInput = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "gradInput", "torch.CudaTensor");
  
  luaL_argcheck(L, input->nDimension == 2, 2, "2D(batch mode) tensor expected");
  luaL_argcheck(L, input->size[1] == inputSize, 2, "invalid input size");  
  
  luaL_argcheck(L, gradOutput->nDimension == 1, 2, "1D tensor expected");
  
  THCudaTensor_resizeAs(gradInput, input);
  
  /* call cudakernel */
  dim3 blocks(input->size[0]); // each block is an example
  dim3 threads(SOFTMAXTREE_THREADS);
  cunnx_SoftMaxTree_updateGradInput_kernel<<<blocks,threads>>>(
    THCudaTensor_data(gradInput), THCudaTensor_data(logsoftOutput), 
    THCudaTensor_data(gradOutput), THCudaTensor_data(weight), 
    THCudaTensor_data(target), THCudaTensor_data(childParent), 
    THCudaTensor_data(parentChildren), 
    input->size[1], rootId, maxFamilyPath
  );
  
  hipError_t errcode = hipGetLastError();
  if(errcode != hipSuccess)
    THError(hipGetErrorString(errcode));
  
  return 1;
}

__global__ void cunnx_SoftMaxTree_accGradParameters_kernel(
  float *gradWeight, float *gradBias, float *input, float* linearGradOutput,
  float *target, float* childParent, float* parentChildren, 
  float scale, int nInput, int rootId, int maxFamilyPath)
{
  __shared__ float buffer[SOFTMAXTREE_THREADS+1];
  int tx = threadIdx.x;
  int i_step = blockDim.x;
  int k = blockIdx.x;
  float *input_k = input + k*nInput;
  float *nodeGradOutput, *nodeGradWeight, *nodeGradBias;
  int childId = target[k] - 1;
  int parentId, parentIdx, childIdx, nChildren;
  float *node;
  int n = 0;
  THIntTensor *node;
  
  // loop through nodes
  while(1)
  {
    /* get next Node in Tree */
    node = childParent + childId*2;
    parentId = (int)node[0] - 1;
    childIdx = (int)node[1] - 1;
    
    node = parentChildren + parentId*2;
    parentIdx = (int)node[0] - 1;
    nChildren = (int)node[1];
    
    nodeGradOutput = linearGradOutput + maxFamilyPath*k + n; 
    nodeGradWeight = gradWeight + parentIdx*nInput;
    nodeGradBias = gradBias + parentIdx;
      
    THTensor_(addr)(nodeGradWeight, 1, nodeGradWeight, scale, nodeGradOutput, nodeInput);
    THTensor_(cadd)(nodeGradBias, nodeGradBias, scale, nodeGradOutput);
    
    // addr weights (scalar-products)
    for (int i=tx; i<nInput; i+=i_step)
    {
      // copy input to buffer
      buffer[tx] = input_k[i];
    
      for (int j=0; j<nChildren; j++)
      {
        // multiply accumulate weights
        nodeGradWeight[j*nInput + i] += scale*nodeGrad[j]*buffer[tx];
      }
    }
    
    // cadd bias
    for (int i=tx; i<nChildren; i+=i_step)
    {
      // multiply accumulate weights
      nodeGradBias[i] += scale*nodeGrad[i]
    }
    
    n += nChildren;
    /* Break when root is reached */
    if (parentId == rootId)
    {
      break;
    }
    childId = parentId;
  }
}

static int cunnx_SoftMaxTree_accGradParameters(lua_State *L)
{
  THCudaTensor *input = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");  
  THCudaTensor *target = (THCudaTensor*)luaT_checkudata(L, 4, "torch.CudaTensor");  
  float scale = luaL_optnumber(L, 5, 1);
  int inputSize = luaT_getfieldcheckint(L, 1, "inputSize");
  int rootId = luaT_getfieldcheckint(L, 1, "rootId") - 1;
  int maxFamilyPath = (int)luaT_getfieldcheckint(L, 1, "maxFamilyPath");
  
  THCudaTensor *childParent = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "childParent", "torch.CudaTensor");
  THCudaTensor *parentChildren = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "parentChildren", "torch.CudaTensor");
  
  THCudaTensor *linearGradOutput = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "_multiBuffer", "torch.CudaTensor");
  
  THCudaTensor *gradWeight = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "gradWeight", "torch.CudaTensor");
  THCudaTensor *gradBias = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "gradBias", "torch.CudaTensor");
  
  lua_getfield(L, 1, "updates"); // this will be a pain to fill
  
  luaL_argcheck(L, input->nDimension == 2, 2, "2D(batch mode) tensor expected");
  luaL_argcheck(L, input->size[1] == inputSize, 2, "invalid input size");  
  
  /* call cudakernel */
  dim3 blocks(input->size[0]); // each block is an example
  dim3 threads(SOFTMAXTREE_THREADS);
  cunnx_SoftMaxTree_accGradParameters_kernel<<<blocks,threads>>>(
    THCudaTensor_data(gradWeight), THCudaTensor_data(Bias), 
    THCudaTensor_data(input), THCudaTensor_data(linearGradOutput), 
    THCudaTensor_data(target), THCudaTensor_data(childParent), 
    THCudaTensor_data(parentChildren), 
    input->size[1], rootId, maxFamilyPath, scale
  );
    
  return 0;
}

static const struct luaL_Reg cunnx_SoftMaxTree__ [] = {
  {"SoftMaxTree_updateOutput", cunnx_SoftMaxTree_updateOutput},
  {"SoftMaxTree_updateGradInput", cunnx_SoftMaxTree_updateGradInput},
  {"SoftMaxTree_accGradParameters", cunnx_SoftMaxTree_accGradParameters},
  {NULL, NULL}
};

static void cunnx_SoftMaxTree_init(lua_State *L)
{
  luaT_pushmetatable(L, "torch.CudaTensor");
  luaT_registeratname(L, cunnx_SoftMaxTree__, "nn");
  lua_pop(L,1);
}
